// 2025 Istituto per le Applicazioni del Calcolo "Mauro Picone"
// alessandro.celestini@cnr.it

#include "gPower.h"

using namespace std;

// Max number of samples per device
#define SAMPLE_MAX_SIZE_DEFAULT 2000000
// Max number of sampled devices
#define MAX_DEVICES 8
// Sampling frequency in seconds
#define TIME_STEP 0.00001

// Flag variable used to end the sampling thread
int terminate_thread = 0;
// NVML handlers
nvmlDevice_t nvDevice[MAX_DEVICES];
// Arrays of sampling timestamp in us 
double thread_times[MAX_DEVICES][SAMPLE_MAX_SIZE_DEFAULT];
// Arrays of power sampling in milliwatts
double thread_powers[MAX_DEVICES][SAMPLE_MAX_SIZE_DEFAULT];
// Total number of samples
int n_values;

// Sampling thread handler
pthread_t thread_sampler;
// Number of devices to monitor
unsigned int device_count;
// Output directory
string glb_out_dir;
// Hostname of the computing node
string glb_node_name;
// File with the starting sample timestamp of each device 
string glb_file_starttime;



void *threadWork(void * arg) {
    unsigned int power[MAX_DEVICES];
    int i=0;
    bool not_enough=0;
    struct timeval tv_aux;
    struct timeval tv_start = *((struct timeval *) arg);
    nvmlReturn_t nvResult;

    printf("*** Start Sampling Thread ***\n");

    while (!terminate_thread) {
	for(int d=0; d < device_count; d++){		       
            nvResult = nvmlDeviceGetPowerUsage(nvDevice[d], &power[d]);
	    if (NVML_SUCCESS != nvResult) {
		printf("Failed to get power usage: %s [device %d]\n", nvmlErrorString(nvResult), d);
		if (nvResult == NVML_ERROR_UNINITIALIZED){
		    printf("NVML_ERROR_UNINITIALIZED: the library has not been successfully initialized\n");
		    pthread_exit(NULL);
		}
		if (nvResult == NVML_ERROR_INVALID_ARGUMENT){
		    printf("NVML_ERROR_INVALID_ARGUMENT: device is invalid or power is NULL\n");
		}
		if (nvResult == NVML_ERROR_NOT_SUPPORTED){
		    printf("NVML_ERROR_NOT_SUPPORTED: the device does not support power readings\n");
		}
		if (nvResult == NVML_ERROR_GPU_IS_LOST){
		    printf("NVML_ERROR_GPU_IS_LOST: the target GPU has fallen off the bus or is otherwise inaccessible\n");
		}
		if (nvResult == NVML_ERROR_UNKNOWN){
		    printf("NVML_ERROR_UNKNOWN on any unexpected error\n");
		}
		if ( i>0 ) {
		  // If a critical error DOES NOT occure we keep the last measure	
		  power[d] = thread_powers[d][i-1];
		}else{
		  power[d] = 0;
		}
	    }			
	    if(i < SAMPLE_MAX_SIZE_DEFAULT ) {
		gettimeofday(&tv_aux,NULL);
            	thread_powers[d][i] = power[d];			
		thread_times[d][i] = (tv_aux.tv_sec-tv_start.tv_sec)*1000000;
            	thread_times[d][i] += (tv_aux.tv_usec-tv_start.tv_usec);
		if(i==0) { 
		    printf("*** Start Power Sampling (device %d) ***\n",d);
		}
		if(i==0) { 
	            FILE *fp_starttime;
		    fp_starttime = fopen(glb_file_starttime.c_str(), "a");
		    fprintf(fp_starttime,"%d;%ld;%ld\n", d, tv_aux.tv_sec, tv_aux.tv_usec);
		    fclose(fp_starttime);
		}
	     }
	     else{
	        if(i == SAMPLE_MAX_SIZE_DEFAULT) {
		    printf("ERROR: POWER VECTOR SIZE EXCEEDED!\n");
		    pthread_exit(NULL);
		}
		if(!not_enough){
		    printf("NOT ENOUGH POWER!\n");
		    not_enough=1;
		}
	     }	
	}//endfor 
	i++;
	n_values = i;
	sleep(TIME_STEP);
    }//endwhile	
    printf("*** Stop Sampling Thread ***\n");
    pthread_exit(NULL);
}


float DataOutput() {
   float p_average;
   double interval;
   double tot_power;
   float power_peak=0;
   FILE  *fp2;
	
   for(int d=0; d < device_count; d++){
	string s = glb_out_dir+"/nvmlPowerProfile_"+std::to_string(d)+"_"+glb_node_name+".csv";
	fp2 = fopen(s.c_str(), "w+");
	fprintf(fp2,"Timestamp [s];Power measure [W]");

	for(int i=0; i<n_values; i++) {
            fprintf(fp2, "\n%.6f;%.4f", (thread_times[d][i]-thread_times[d][0])/1000000, thread_powers[d][i]/1000.0);	
            if (thread_powers[d][i] > power_peak){ 
        	power_peak = thread_powers[d][i];
	    }
	    tot_power += thread_powers[d][i];
	}
	fclose(fp2);
   }

   p_average = tot_power/(device_count*n_values);    
   interval = thread_times[0][n_values-1] - thread_times[0][0];

   printf("\tAverage Power: %.2f W;  Max Power: %.2f W;  Sampling Duration: %.2f s;  Samples number: %d\n", p_average/1000.0, power_peak/1000.0, (interval)/1000000, n_values);
   
   return 0;
}

int GPowerU_init(string out_dir, string node_name) {	
   struct timeval start_time;

   gettimeofday(&start_time,NULL);
   glb_out_dir = out_dir;
   glb_node_name = node_name;
   nvmlReturn_t nvResult;

   if ( mkdir(glb_out_dir.c_str(), 0777) < 0 && errno != EEXIST){
      printf("Unable to create the output directory named: %s", glb_out_dir);
      exit(-1); 
   }

   FILE *fp_starttime;
   glb_file_starttime = glb_out_dir + "/startTime_"+ glb_node_name +".time";
   fp_starttime = fopen(glb_file_starttime.c_str(), "w");
   fprintf(fp_starttime,"device;tv_sec;tv_usec\n");
   fclose(fp_starttime);
    
   terminate_thread = 0;
   nvResult = nvmlInit();
   if (NVML_SUCCESS != nvResult){
        printf("Failed to initialize NVML: %s\n", nvmlErrorString(nvResult));
        printf("Press ENTER to continue...\n");
        getchar();
        return -1;
   }
   nvResult = nvmlDeviceGetCount(&device_count);
   if (NVML_SUCCESS != nvResult){
        printf("Failed to query device count: %s\n", nvmlErrorString(nvResult));
        return -1;
   }
   printf("Found %d device%s\n\n", device_count, device_count != 1 ? "s" : "");
   if (device_count > MAX_DEVICES) {
        printf("Device_id is out of range.\n");
        return -1;
   }

   for(int d=0; d<device_count; d++){
     nvResult = nvmlDeviceGetHandleByIndex(d, &nvDevice[d]);
     if (NVML_SUCCESS != nvResult){
       printf("Failed to get handle for device %d: %s\n",d, nvmlErrorString(nvResult));
       return -1;
     }
   }
   int a = pthread_create(&thread_sampler, NULL, threadWork, &start_time);
   if(a) {
     fprintf(stderr,"Error - pthread_create() return code: %d\n",a);
     return -1;
   }

   return 0;
}


int GPowerU_end() {
   terminate_thread = 1;
   pthread_join(thread_sampler, NULL);
   DataOutput();
   return 0;
}
