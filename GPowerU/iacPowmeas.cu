#include <iostream>
#include <math.h>
#include "nvml.h"
#include "hip/hip_runtime_api.h"
#include "GPowerU.hpp"
#include <unistd.h>
#include <stdio.h>
#include <signal.h>     


static void sigHandler(int signum){
    if (signum == SIGINT){
        fprintf ( stderr, "SIGINT catched!\n" );
        printf("Ready to exit!\n");
        if ( GPowerU_end(5) != 0 ){
            fprintf ( stderr, " error: terminating...\n" );
            _exit (1);
        }
        printf("Done!\n");
        _exit (0);
    }
}
 

int main( int argc, char** argv){

    //Initializations ==> enable the NVML library, starts CPU thread for the power monitoring,  
    if ( GPowerU_init() != 0 ) {
        fprintf ( stderr, "%s: error: initializing...\n", argv[0] );
        _exit (1);
     }

    if (signal(SIGINT, sigHandler) == SIG_ERR){
        printf("Error, cannot handle SIGINT\n");
    }

    printf("Start FOR\n");
    for(;;){
       sleep(30);
    }
    fprintf( stderr, "ERROR: End FOR!!!\n");

    return 0;
}

