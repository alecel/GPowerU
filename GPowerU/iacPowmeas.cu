#include <iostream>
#include <math.h>
#include "nvml.h"
#include "hip/hip_runtime_api.h"
#include "GPowerU.hpp"
#include <unistd.h>
#include <stdio.h>
#include <signal.h>     
#include <getopt.h>

#define USAGE                                                                                          \
    "Usage: %s --output-dir <DIR_NAME> \n\n"                                                           \
    "\t-o, --output-dir <DIR>              Write power profile data files to <DIR_NAME>.\n\n"



static void sigHandler(int signum){
    if (signum == SIGINT){
        fprintf( stderr, "SIGINT catched!\n" );
        fprintf( stderr, "Ready to exit!\n");
        if ( GPowerU_end(5) != 0 ){
            fprintf ( stderr, " error: terminating...\n" );
            _exit (1);
        }
        printf("Done!\n");
        _exit (0);
    }
}
 

int main( int argc, char** argv){
   
  char node_name[256];	
  char *out_dir=NULL;
  int opt;	
  static struct option long_options[] = {
        { "output-dir", required_argument, NULL, 'o' }
  };

  while ((opt = getopt_long(argc, argv, "o:h", long_options, NULL)) != -1) {
    switch (opt) {
     case 'o':
       out_dir = strdup(optarg);
       break;
     case 'h':
     default:
       printf(USAGE, argv[0]);
       exit(EXIT_FAILURE);
    }
  }
  
  if ( out_dir == NULL){
    printf(USAGE, argv[0]);
    exit(EXIT_FAILURE);
  }

  memset(node_name, 0, 256);
  gethostname(node_name, 256);
  
  printf("out_dir: %s --- node_name: %s \n", out_dir, node_name);

  //Initializations ==> enable the NVML library, starts CPU thread for the power monitoring,  
  if ( GPowerU_init(out_dir, node_name) != 0 ) {
    fprintf ( stderr, "%s: error: initializing...\n", argv[0] );
    _exit (1);
  }

  if (signal(SIGINT, sigHandler) == SIG_ERR){
    printf("Error, cannot handle SIGINT\n");
  }

  for(;;){
    sleep(30);
  }

  return 0;
}

